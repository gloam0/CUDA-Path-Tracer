#include "hip/hip_runtime.h"
#include "exr_utils.cuh"

#include <iostream>
#include <thread>

#include <hip/hip_runtime.h>

#include "cuda_utils.cuh"
#include "common.cuh"

#define TINYEXR_IMPLEMENTATION
#include "tinyexr.h"

////////////////////////////////////////////////////////////////////////////////////////////////
void create_env_map_threaded(const char* filepath, hdr_map* map, bool* success) {
    std::thread tmp(create_env_map, filepath, map, success);
    tmp.detach();
}
////////////////////////////////////////////////////////////////////////////////////////////////
void create_env_map(const char* filepath, hdr_map* map, bool* success) {
    float* exr_data;
    int width, height;
    const char* err = nullptr;

    int ret = LoadEXR(&exr_data, &width, &height, filepath, &err);
    if (ret != TINYEXR_SUCCESS) {
        if (err) {
            std::cerr << "Error loading EXR file: " << err << std::endl;
            FreeEXRErrorMessage(err);
        } else {
            std::cerr << "Error loading EXR file." << std::endl;
        }
        *success = false;
        hipMemcpyToSymbol(HIP_SYMBOL(d_use_hdr), success, sizeof(bool));
        return;
    }

    map->width = width;
    map->height = height;

    /* Create array on device and copy EXR data */
    hipChannelFormatDesc channel_desc = hipCreateChannelDesc<float4>();
    CHECK_ERR(hipMallocArray(&map->cuda_array, &channel_desc, width, height));
    CHECK_ERR(hipMemcpy2DToArray(map->cuda_array, 0, 0, exr_data, width * sizeof(float4),
        width * sizeof(float4), height, hipMemcpyHostToDevice));

    hipResourceDesc res_desc = {};
    res_desc.resType = hipResourceTypeArray;
    res_desc.res.array.array = map->cuda_array;

    /* Describe and create a texture object for the EXR data */
    hipTextureDesc tex_desc = {};
    tex_desc.addressMode[0] = hipAddressModeWrap;  /* Wrap horizontally */
    tex_desc.addressMode[1] = hipAddressModeWrap;  /* Wrap vertically */
    tex_desc.filterMode = hipFilterModePoint;      /* Point sampling, faster than linear */
    tex_desc.readMode = hipReadModeElementType;    /* Read as specified type */
    tex_desc.normalizedCoords = 1;                  /* Use normalized coords */
    CHECK_ERR(hipCreateTextureObject(&map->tex_obj, &res_desc, &tex_desc, nullptr));

    *success = true;
    env_map_loaded = true;
    hipMemcpyToSymbol(HIP_SYMBOL(d_use_hdr), success, sizeof(bool));
}
////////////////////////////////////////////////////////////////////////////////////////////////
void free_env_map(hdr_map map) {
    hipDestroyTextureObject(map.tex_obj);
    hipFreeArray(map.cuda_array);
}
////////////////////////////////////////////////////////////////////////////////////////////////
