#include "hip/hip_runtime.h"
#include "camera.cuh"

#include <hip/hip_runtime.h>

#include "common.hpp"
#include "cuda_utils.cuh"

////////////////////////////////////////////////////////////////////////////////////////////////
__device__ ray3 make_ray(float vp_i, float vp_j) {
    vec3 vp_point = d_camera_params.vp_start + vp_i * d_camera_params.vp_delta_px_w
                                             + vp_j * d_camera_params.vp_delta_px_h;
    vec3 direction = normalize(vp_point - d_camera_params.c_location);
    return ray3{
        d_camera_params.c_location,
        direction
    };
}
////////////////////////////////////////////////////////////////////////////////////////////////
void Camera::init() {
    h_camera_params.c_location = view::init_camera_loc;
    h_camera_params.c_direction = normalize(view::init_camera_dir);

    h_camera_params.c_yaw = 0.0f;
    h_camera_params.c_pitch = 0.0f;

    h_camera_params.c_dlat = 0.f;
    h_camera_params.c_dlong = 0.f;

    quaternion yaw_rotation = make_rotor_quaternion(view::init_camera_up, h_camera_params.c_yaw * deg_to_rad);
    quaternion pitch_rotation = make_rotor_quaternion(view::init_camera_right, h_camera_params.c_pitch * deg_to_rad);
    h_camera_params.c_orientation = yaw_rotation * pitch_rotation;
    h_camera_params.c_orientation = normalize(h_camera_params.c_orientation);

    h_camera_params.focal_length = view::init_focal_length;

    update();
}
////////////////////////////////////////////////////////////////////////////////////////////////
void Camera::update() {
    if (!h_input_state.free_mode) return;

    vec3 c_direction = rotate_v(view::init_camera_dir, h_camera_params.c_orientation);
    vec3 c_up = rotate_v(normalize(view::init_camera_up), h_camera_params.c_orientation);
    vec3 c_right = rotate_v(view::init_camera_right, h_camera_params.c_orientation);

    h_camera_params.c_direction = c_direction;

    h_camera_params.vp_w_vec = view::w * c_right;
    h_camera_params.vp_h_vec = view::h * c_up;

    h_camera_params.vp_delta_px_w = h_camera_params.vp_w_vec / img::w;
    h_camera_params.vp_delta_px_h = h_camera_params.vp_h_vec / img::h;

    h_camera_params.vp_start =
        h_camera_params.c_location                                              /* camera loc */
      + h_camera_params.focal_length * h_camera_params.c_direction              /* vector from camera to vp center */
      - 0.5 * (h_camera_params.vp_w_vec + h_camera_params.vp_h_vec)             /* center to top left vp */
      + 0.5 * (h_camera_params.vp_delta_px_w + h_camera_params.vp_delta_px_h);  /* to px center (1/2 px dimensions) */

    CHECK_ERR(hipMemcpyToSymbol(HIP_SYMBOL(d_camera_params), &h_camera_params, sizeof(CameraParams)));
}
////////////////////////////////////////////////////////////////////////////////////////////////
void Camera::frame_now(double frame_time) {
    if (!h_input_state.free_mode) return;

    double dt = frame_time / 1000000;
    /* Apply acceleration based on key states */
    if (wasd_state.W)
        h_camera_params.c_dlong += (view::move_accel_scale + view::move_decel_scale) * dt;

    if (wasd_state.S)
        h_camera_params.c_dlong -= (view::move_accel_scale + view::move_decel_scale) * dt;

    if (wasd_state.A)
        h_camera_params.c_dlat -= (view::move_accel_scale + view::move_decel_scale) * dt;

    if (wasd_state.D)
        h_camera_params.c_dlat += (view::move_accel_scale + view::move_decel_scale) * dt;


    /* Apply deceleration when keys are not pressed */
    if (h_camera_params.c_dlong > 0.f) {
        h_camera_params.c_dlong -= view::move_decel_scale * dt;
        if (h_camera_params.c_dlong < 0.f) h_camera_params.c_dlong = 0.f;
    } else if (h_camera_params.c_dlong < 0.f) {
        h_camera_params.c_dlong += view::move_decel_scale * dt;
        if (h_camera_params.c_dlong > 0.f) h_camera_params.c_dlong = 0.f;
    }

    if (h_camera_params.c_dlat > 0.f) {
        h_camera_params.c_dlat -= view::move_decel_scale * dt;
        if (h_camera_params.c_dlat < 0.f) h_camera_params.c_dlat = 0.f;
    } else if (h_camera_params.c_dlat < 0.f) {
        h_camera_params.c_dlat += view::move_decel_scale * dt;
        if (h_camera_params.c_dlat > 0.f) h_camera_params.c_dlat = 0.f;
    }

    if (h_camera_params.c_dlong > view::max_velocity) h_camera_params.c_dlong = view::max_velocity;
    if (h_camera_params.c_dlong < -view::max_velocity) h_camera_params.c_dlong = -view::max_velocity;

    if (h_camera_params.c_dlat > view::max_velocity) h_camera_params.c_dlat = view::max_velocity;
    if (h_camera_params.c_dlat < -view::max_velocity) h_camera_params.c_dlat = -view::max_velocity;

    /* Update position based on velocity and delta time */
    vec3 c_forward = normalize(h_camera_params.c_direction);
    vec3 c_right = normalize(rotate_v(view::init_camera_right, h_camera_params.c_orientation));

    /* Update position based on velocity and delta time */
    h_camera_params.c_location +=
        c_forward * float(h_camera_params.c_dlong * dt) +
        c_right * float(h_camera_params.c_dlat * dt);

    update();
}
////////////////////////////////////////////////////////////////////////////////////////////////
void Camera::scroll_zoom(float d_focal_length) {
    if (!h_input_state.free_mode) return;

    h_camera_params.focal_length += d_focal_length;

    h_camera_params.focal_length = h_camera_params.focal_length < view::min_focal_length ? view::min_focal_length : h_camera_params.focal_length;
    h_camera_params.focal_length = h_camera_params.focal_length > view::max_focal_length ? view::max_focal_length : h_camera_params.focal_length;
}
////////////////////////////////////////////////////////////////////////////////////////////////
void Camera::mouse_rotate(const float2& mouse_delta) {
    if (!h_input_state.free_mode) return;

    /* Convert mouse movement to rotation angles */
    float delta_yaw = mouse_delta.x * view::look_sensitivity;
    float delta_pitch = mouse_delta.y * view::look_sensitivity;

    h_camera_params.c_yaw += delta_yaw;
    h_camera_params.c_pitch += delta_pitch;

    /* Create rotation quaternions for yaw and pitch rotations */
    quaternion yaw_rotation = make_rotor_quaternion(view::init_camera_up, h_camera_params.c_yaw * deg_to_rad);
    quaternion pitch_rotation = make_rotor_quaternion(view::init_camera_right, h_camera_params.c_pitch * deg_to_rad);

    /* Combine rotations */
    h_camera_params.c_orientation = normalize(yaw_rotation * pitch_rotation);

    /* Call update to recalculate other parameters */
    update();
}
////////////////////////////////////////////////////////////////////////////////////////////////
Camera::wasdState Camera::wasd_state;
////////////////////////////////////////////////////////////////////////////////////////////////
