#include "hip/hip_runtime.h"
#include "render.cuh"

#include <cuda_gl_interop.h>
#include <GLFW/glfw3.h>

#include "common.hpp"
#include "cuda_utils.cuh"
#include "input.hpp"
#include "camera.cuh"
#include "hit.cuh"
#include "material.cuh"

////////////////////////////////////////////////////////////////////////////////////////////////
__global__ void init_curand(hiprandState *curand_state, unsigned long long seed) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int k = blockIdx.z * blockDim.z + threadIdx.z;

    int idx = i + j * (blockDim.x * gridDim.x) + k * (blockDim.x * gridDim.x * blockDim.y * gridDim.y);

    hiprand_init(seed, idx, 0, &curand_state[idx]);
}
////////////////////////////////////////////////////////////////////////////////////////////////
void render_frame(scene* d_scene, hipGraphicsResource* pbo_resource, int render_mode_frame_count) {
    /* Map the shared pbo to allow writing and get a pointer to it */
    uchar4* cuda_pbo;
    size_t num_bytes;
    hipGraphicsMapResources(1, &pbo_resource, 0);
    hipGraphicsResourceGetMappedPointer((void**)&cuda_pbo, &num_bytes, pbo_resource);

    /* Render the current frame with multisampling and reduce multisamples */
    render_scene<<<grid_size, block_size>>>(cuda_pbo, d_scene, render_mode_frame_count);
    CHECK_ERR(hipGetLastError());

    /* Unmap the shared pbo to finish writing and synchronize */
    hipGraphicsUnmapResources(1, &pbo_resource, 0);

    /* Update the texture
     *    GL_PIXEL_UNPACK_BUFFER is bound (to the pbo) and will be used to update
     *    the texture, hence *pixels is used as an offset which we set to 0. */
    glTexSubImage2D(GL_TEXTURE_2D, 0, 0, 0, img::w, img::h, GL_RGBA, GL_UNSIGNED_BYTE, 0);

    /* Draw the texture to the fullscreen quad */
    glClear(GL_COLOR_BUFFER_BIT);
    glEnable(GL_TEXTURE_2D);
    glBegin(GL_QUADS);
    glTexCoord2f(0, 0); glVertex2f(-1, -1);
    glTexCoord2f(1, 0); glVertex2f(1, -1);
    glTexCoord2f(1, 1); glVertex2f(1, 1);
    glTexCoord2f(0, 1); glVertex2f(-1, 1);
    glEnd();
    glDisable(GL_TEXTURE_2D);

    /* Swap back/front buffers, handled by GLFW */
    glfwSwapBuffers(glfwGetCurrentContext());

    /* Handle events that occurred this frame (see callbacks in input.h) */
    glfwPollEvents();
}
////////////////////////////////////////////////////////////////////////////////////////////////
__global__ void check_borders(uchar4 *out) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i < img::w && j < img::h) {
        int index = j * img::w + i;

        unsigned char c = (i == 0) || (i == img::w-1) || (j == 0) || (j == img::h-1);
        out[index] = make_uchar4(c * 255, c * 255, c * 255, 255);
    }
}
////////////////////////////////////////////////////////////////////////////////////////////////
__global__ void render_scene(uchar4* out, scene* s, int render_mode_frame_count) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i >= img::w || j >= img::h ) return;

    int thread_index = j * img::w + i;
    unsigned int seed = d_randoms[thread_index];

    /* Get a random subpixel in the pixel associated with this thread */
    auto sub_x = i + xorshift32_f_norm(&seed);
    auto sub_y = j + xorshift32_f_norm(&seed);

    /* Trace this ray and write its resulting color to the multisampling buffer */
    float3 col = trace_ray(sub_x, sub_y, s, &seed);

    if (d_input_state.free_mode) {
        /* Free mode: write frame directly to out buffer */
        out[thread_index] = make_uchar4(
            (unsigned char)(255.999f * gamma_correct(col.x)),
            (unsigned char)(255.999f * gamma_correct(col.y)),
            (unsigned char)(255.999f * gamma_correct(col.z)),
            255
        );
    } else {
        /* Render mode */
        /* clear render_mode_buffer on first frame in render mode */
        if (d_input_state.render_mode_first_frame) {
            d_render_mode_buff[thread_index] = make_float4(0.f, 0.f, 0.f, 0.f);
        }
        /* Accumulate color */
        d_render_mode_buff[thread_index] += make_float4(col.x, col.y, col.z, 255);
        /* Write current color average to out buffer */
        float inv_frame_count = 1.0f / render_mode_frame_count;
        out[thread_index] = make_uchar4(
            (unsigned char)(255.999f * gamma_correct(d_render_mode_buff[thread_index].x
                                     * inv_frame_count)),
            (unsigned char)(255.999f * gamma_correct(d_render_mode_buff[thread_index].y
                                     * inv_frame_count)),
            (unsigned char)(255.999f * gamma_correct(d_render_mode_buff[thread_index].z
                                     * inv_frame_count)),
            255
        );
    }

    /* Store random state */
    d_randoms[thread_index] = xorshift32_i(&seed);
}
////////////////////////////////////////////////////////////////////////////////////////////////
__device__ color3 trace_ray(float vp_x, float vp_y, scene* s, unsigned int* seed) {
    ray3 r = make_ray(vp_x, vp_y);      /* Get ray through (vp_x, vp_y) */
    color3 curr_attenuation = color3{1.f,1.f,1.f};   /* Attenuation */

    hit best_hit;
    float best_t;
    hit this_hit;
    int depth = 0;
    do {  /* check scene objects for intersection, track nearest hit */
        best_t = -1.f;
        for (int idx = 0; idx < s->num_objects; idx++){
            if (s->spheres[idx].is_hit(r, this_hit)) {
                if (this_hit.t < best_t || best_t < 0) {
                    best_hit = this_hit;
                    best_hit.mat = &s->materials[idx];
                    best_t = this_hit.t;
                }
            }
        }
        if (best_t > 0.f) {
            /* scatter r and apply attenuation */
            curr_attenuation = elem_product(curr_attenuation,
                                            scatter(&r,&best_hit, best_hit.mat, seed));
        } else {
            /* No hit, use background color and exit */
            return elem_product(curr_attenuation, render::background_color);
        }
        depth++;
    } while (depth < render::max_scatter_depth);

    return curr_attenuation;
}